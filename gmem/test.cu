#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int* a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = 1;
    }
}

__global__ void test(int* da, int* dr, int N) {
    int tid = (blockDim.x * blockIdx.x + threadIdx.x) * 16;
    for (; tid < N; tid += blockDim.x * gridDim.x * 16)
        int a = da[tid];
}

int main() {
    const int N = 1024 * 1024 * 1024;

    int* a = new int[N];
    int* da;
    init(a, N);
    hipMalloc(&da, sizeof(int) * N);
    hipMemcpy(da, a, sizeof(int) * N, hipMemcpyHostToDevice);

    int* r = new int[N];
    int* dr;
    hipMalloc(&dr, sizeof(int) * N);

    test<<<1024, 1024>>>(da, dr, N);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}