#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

void init(float* a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = 1;
    }
}
__global__ void readglobal(float* da, float* db, int N) {
    int x = threadIdx.x;
    reinterpret_cast<float2*>(db)[x] = reinterpret_cast<float2*>(da)[x];
}

int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);
    const int N = 128;
    float* a = new float[N];
    float *da, *db;
    init(a, N);
    hipMalloc(&da, sizeof(float) * N);
    hipMalloc(&db, sizeof(float) * N);
    hipMemcpy(da, a, sizeof(float) * N, hipMemcpyHostToDevice);

    readglobal<<<1, 32>>>(da, db, N);

    hipMemcpy(a, da, sizeof(float) * N, hipMemcpyDeviceToHost);
}