#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
using namespace std;

__global__ void baseline(int* a) {
    long long start, stop;
    asm volatile(
        "{\n\t"
        "mov.u64 %0, %%clock64;\n\t"
        "mov.s32 %1, 1;\n\t"
        "mov.u64 %2, %%clock64;\n\t"
        "}"
        : "=l"(start), "=r"(*a), "=l"(stop));

    printf("%lld\n", stop - start);
}

int main() {
    int dev = 3;
    hipSetDevice(dev);
    int thread_num_per_block = 1;
    int block_num = 1;
    int ha = 0;
    int* da;
    hipMalloc(&da, sizeof(int));
    hipMemcpy(da, &ha, sizeof(int), hipMemcpyHostToDevice);
    dim3 block(thread_num_per_block);
    dim3 grid(block_num);
    baseline<<<grid, block>>>(da);
    //cudaDeviceSynchronize();
    hipMemcpy(&ha, da, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d", ha);
}