
#include <hip/hip_runtime.h>
#include<iostream>
const int N = 1 << 10;

__global__ void kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main() {
    float *data;
    float *d_data;
    hipHostMalloc(&data, N * sizeof(float), hipHostMallocDefault);
    for(int i=0;i<10;i++){
        data[i] = i;
    }
    hipMalloc(&d_data,N*sizeof(float));
    hipMemcpy(d_data,data,sizeof(float)*N,hipMemcpyHostToDevice);
    hipStream_t cur;
    hipStreamCreate(&cur);
    for (int i = 0; i < 1; i++) {
        kernel<<<1, 64, 0, cur>>>(d_data, N);
    }
    hipMemcpy(data,d_data,sizeof(float)*N,hipMemcpyDeviceToHost);
    for(int i=0;i<10;i++){
        std::cout<<data[i];
    }
    return 0;
}

// int main() {
//     cudaGraph_t graph;
//     cudaGraphExec_t instance;


//     float *data;
//     cudaMalloc(&data, N * sizeof(float));
//     cudaStream_t cur;
//     cudaStreamCreate(&cur);

//     cudaStreamBeginCapture(cur,cudaStreamCaptureModeGlobal);
//     for (int i = 0; i < 10; i++) {
//         kernel<<<1, 64, 0, cur>>>(data, N);
//     }
//     cudaStreamEndCapture(cur,&graph);
//     cudaGraphInstantiate(&instance,graph,0);

//     cudaGraphLaunch(instance,cur);
//     return 0;
// }
