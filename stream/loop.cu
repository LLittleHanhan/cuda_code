
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
const int N = 1 << 16;

__global__ void kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main() {
    hipStream_t s1,s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    hipEvent_t e1,e2;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    
    float *data;
    data = new float[N];
    
    float *d_data;
    hipMalloc(&d_data,N*sizeof(float));
    hipMemcpy(d_data,data,sizeof(float)*N,hipMemcpyHostToDevice);
 
    for(int i=0;i<10;i++){
        kernel<<<1, 64, 0, s1>>>(d_data, N);
      
            hipEventRecord(e1,s1);
        hipStreamWaitEvent(s2,e1);
        kernel<<<1, 64, 0, s2>>>(d_data, N/2);
    }
    
    

    hipMemcpy(data,d_data,sizeof(float)*N,hipMemcpyDeviceToHost);
    for(int i=0;i<10;i++){
        std::cout<<data[i];
    }
    return 0;
}