#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <iostream>
using namespace std;

const int N = 256;
const int thread_num_per_block = 256;
const int block_num = N / (thread_num_per_block);

void initval(float s[], int n) {
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        // s[i] = float(rand() % 10) / 100000;
        s[i] = 0.001;
    }
}

__global__ void scan(float* input) {
    __shared__ float array[thread_num_per_block];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    array[tid] = input[idx];
    __syncthreads();

    for (int i = 1; i <= thread_num_per_block / 2; i *= 2) {
        if(tid<thread_num_per_block/2){
            array[tid + i + tid / i * i] += array[i - 1 + tid / i * 2 * i];
        }
        __syncthreads();
    }
    input[idx] = array[tid];
    return;
}

int main() {
    int dev = 3;
    hipSetDevice(dev);

    float* a = new float[N];
    float* res = new float[N];
    initval(a, N);

    float* input;
    hipMalloc(&input, sizeof(float) * N);
    hipMemcpy(input, a, sizeof(float) * N, hipMemcpyHostToDevice);
    dim3 block(thread_num_per_block);
    dim3 grid(block_num);

    int iter = 1;
    float msecTotal;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iter; i++) {
        scan<<<grid, block>>>(input);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);

    hipMemcpy(res, input, sizeof(float) * N, hipMemcpyDeviceToHost);

    cout << "sklansky time:" << msecTotal / iter << "ms" << endl;

    // for (int i = 0; i < thread_num_per_block; i++) {
    //     cout << res[i] << " "<<endl;
    // }
    hipFree(input);
    delete[] a;
    delete[] res;
}