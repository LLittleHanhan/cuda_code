#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include <iostream>

using namespace std;

const int warpsize = 32;
const int thread_num_per_block = 128;
const int block_num = 1;
const int N = thread_num_per_block * block_num;

void initval(float s[], int n) {
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        // s[i] = float(rand() % 10) / 100000;
        s[i] = 1;
    }
}

__global__ void baseline(float* input) {
    // Specialize WarpScan for type int
    typedef hipcub::WarpScan<int> WarpScan;

    // Allocate WarpScan shared memory for 4 warps
    __shared__ typename WarpScan::TempStorage temp_storage[4];
    // Obtain one input item per thread
    int tid = threadIdx.x;
    int thread_data = input[blockIdx.x * blockDim.x + tid];

    // Compute inclusive warp-wide prefix sums
    int warp_id = threadIdx.x / 32;
    WarpScan(temp_storage[warp_id]).InclusiveSum(thread_data, thread_data);

    input[blockIdx.x * blockDim.x + tid] = thread_data;
    return;
}

__global__ void scan(float* input) {
    int tid = threadIdx.x;
    int warpLane = tid % warpsize;
    float thread_data = input[blockIdx.x * blockDim.x + tid];

    float temp;

    temp = __shfl_up_sync(0xffffffff, thread_data, 1);
    if (warpLane >= 1)
        thread_data += temp;
    temp = __shfl_up_sync(0xffffffff, thread_data, 2);
    if (warpLane >= 2)
        thread_data += temp;
    temp = __shfl_up_sync(0xffffffff, thread_data, 4);
    if (warpLane >= 4)
        thread_data += temp;
    temp = __shfl_up_sync(0xffffffff, thread_data, 8);
    if (warpLane >= 8)
        thread_data += temp;
    temp = __shfl_up_sync(0xffffffff, thread_data, 16);
    if (warpLane >= 16)
        thread_data += temp;

    input[blockIdx.x * blockDim.x + tid] = thread_data;
    return;
}

int main() {
    int dev = 3;
    hipSetDevice(dev);

    float* a = new float[N];
    float* res = new float[N];
    initval(a, N);

    float* input;
    hipMalloc(&input, sizeof(float) * N);
    hipMemcpy(input, a, sizeof(float) * N, hipMemcpyHostToDevice);
    dim3 block(thread_num_per_block);
    dim3 grid(block_num);

    int iter = 1;
    float msecTotal;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iter; i++) {
        scan<<<grid, block>>>(input);
        baseline<<<grid, block>>>(input);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);

    hipMemcpy(res, input, sizeof(float) * N, hipMemcpyDeviceToHost);

    cout << "warp time:" << msecTotal / iter << "ms" << endl;

    // for (int i = 0; i < N; i++) {
    //     cout << res[i] << endl;
    // }
    hipFree(input);
    delete[] a;
    delete[] res;
}