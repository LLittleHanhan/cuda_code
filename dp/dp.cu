#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ ChildKernel(void* data) {
    printf("in child thread is %d", threadIdx.x);
}

__global__ ParentKernel(void* data, hipStream_t& stream2) {
    ChildKernel<<<1, 32, 0, stream2>>>(data);
    printf("in parent thread is %d", threadIdx.x)
}

int main() {
    hipStream_t stream1;
    hipStreamCreate(&stream1);

    hipStream_t stream2;
    hipStreamCreate(&stream2);
    void* data;
    ParentKernel<<<1, 32, 0, stream1>>>(data, stream2);
}