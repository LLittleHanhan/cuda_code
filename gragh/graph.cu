#include <iostream>

int main() {
    hipGraph_t graph;
    hipGraphExec_t instance;

    float* data;
    hipMalloc(&data, N * sizeof(float));
    hipStream_t cur;
    hipStreamCreate(&cur);

    hipStreamBeginCapture(cur, hipStreamCaptureModeGlobal);
    for (int i = 0; i < 10; i++) {
        kernel<<<1, 64, 0, cur>>>(data, N);
    }
    hipStreamEndCapture(cur, &graph);

    hipGraphInstantiate(&instance, graph, 0);
    hipGraphLaunch(instance, cur);
    return 0;
}
