#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#define abs(x) ((x) >= 0.0 ? (x) : -(x))

void initmatrix(float* s, int M, int N) {
    srand((unsigned)time(NULL));
    for (int i = 0; i < M * N; i++) {
        // s[i * N + j] = float(rand() % 100) / 100;
        s[i] = 1.0;
    }
    return;
}

void check_result(float* a, float* b, float* c, int M, int K, int N) {
    float error = 0;
    float temp = 0;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            temp = 0;
            for (int k = 0; k < K; k++) {
                temp += a[i * K + k] * b[k * N + j];
            }
            if (abs(temp - c[i * N + j]) > error) {
                error = abs(temp - c[i * N + j]);
            }
            // error += abs(temp - c[i * N + j]);
        }
    }
    printf("error is %.8f\n", error);
    return;
}

__global__ void sgemm(float* da, float* db, float* dc, int M, int K, int N) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float temp = 0;

    for (int k = 0; k < K; k++) {
        temp += da[(bx * blockDim.x + tx) * K + k] * db[k * N + by * blockDim.y + ty];
    }
    dc[(bx * blockDim.x + tx) * N + by * blockDim.y + ty] = temp;
    return;
}
/*
argv[1] = M
argv[2] = N
argv[3] = K
*/
int main(int argc, char** argv) {
    int dev = 0;
    hipSetDevice(dev);

    const int M = 1024;
    const int N = 1024;
    const int K = 1024;

    const int tx = 16;
    const int ty = 16;

    float* a = new float[M * K];
    float* b = new float[K * N];
    float* c = new float[M * N];
    initmatrix(a, M, K);
    initmatrix(b, K, N);

    float *da, *db, *dc;
    hipMalloc(&da, sizeof(float) * M * K);
    hipMalloc(&db, sizeof(float) * K * N);
    hipMalloc(&dc, sizeof(float) * M * N);
    hipMemcpy(da, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    dim3 block(tx, ty);
    dim3 grid(M / tx, N / ty);

    sgemm<<<grid, block>>>(da, db, dc, M, K, N);

    hipMemcpy(c, dc, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    check_result(a, b, c, M, K, N);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}