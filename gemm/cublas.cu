#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#define abs(x) ((x) >= 0.0 ? (x) : -(x))

void initmatrix(float* s, int M, int N) {
    srand((unsigned)time(NULL));
    for (int i = 0; i < M * N; i++) {
        // s[i * N + j] = float(rand() % 100) / 100;
        s[i] = i;
    }
    return;
}

void check_result(float* a, float* b, float* c, int M, int K, int N) {
    float error = 0;
    float temp = 0;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            temp = 0;
            for (int k = 0; k < K; k++) {
                temp += a[i * K + k] * b[k * N + j];
            }
            if (abs(temp - c[i * N + j]) > error) {
                error = abs(temp - c[i * N + j]);
            }
            // error += abs(temp - c[i * N + j]);
        }
    }
    printf("error is %.8f\n", error);
    return;
}

/*
argv[1] = M
argv[2] = N
argv[3] = K
*/
int main(int argc, char** argv) {
    int dev = 3;
    hipSetDevice(dev);

    const int M = 1024;
    const int K = 1024;
    const int N = 1024;

    float* a = new float[M * K];
    float* b = new float[K * N];
    float* c = new float[M * N];
    initmatrix(a, M, K);
    initmatrix(b, K, N);

    float *da, *db, *dc;
    hipMalloc(&da, sizeof(float) * M * K);
    hipMalloc(&db, sizeof(float) * K * N);
    hipMalloc(&dc, sizeof(float) * M * N);
    hipMemcpy(da, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1;
    float beta = 0;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, da, K, db, N, &beta, dc, M);
    hipMemcpy(c, dc, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipblasDestroy(handle);
    //check_result(a, b, c, M, K, N);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}