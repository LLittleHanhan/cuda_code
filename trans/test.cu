#include <hip/hip_runtime.h>
#include <iostream>
#include "Profile.h"
using namespace std;

int main() {
    int batch_size = 1 << 20;
    for (; batch_size <= 1 << 30; batch_size *= 2) {
        cout << "batch size is " << batch_size / 1 << 20 << " MB" << endl;
        float* a = new float[batch_size];
        float* da;
        hipMalloc(&da, sizeof(float) * batch_size);
        {
            Profiler p("");
            hipMemcpy(da, a, sizeof(float) * batch_size, hipMemcpyHostToDevice);
        }
        delete[] a;
        hipFree(da);
    }
}