/*
展开最后一个线程束
*/
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdlib.h>
using namespace std;

const int N = 256 * 1024 * 128;
const int m = 2;
const int thread_num_per_block = 256;
const int block_num = N / (thread_num_per_block * m);

void initval(float s[], int n)
{
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++)
    {
        s[i] = float(rand() % 10) / 100000;
        // s[i] = 1.0;
    }
}

__global__ void reduce(float *input, float *output)
{
    __shared__ float array[thread_num_per_block];
    int tid = threadIdx.x;
    int id = blockIdx.x * blockDim.x * m + threadIdx.x;
    array[tid] = input[id] + input[id + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 32; s /= 2)
    {
        if (tid < s)
        {
            array[tid] += array[tid + s];
        }
        __syncthreads();
    }
    if (tid < 32)
    {
        array[tid] += array[tid + 32];
        array[tid] += array[tid + 16];
        array[tid] += array[tid + 8];
        array[tid] += array[tid + 4];
        array[tid] += array[tid + 2];
        array[tid] += array[tid + 1];
    }
    if (tid == 0)
    {
        output[blockIdx.x] = array[0];
    }
}

int main(int argc, char **argv)
{
    int dev = 0;
    hipSetDevice(dev);

    float *a = new float[N];
    float *res = new float[block_num];
    initval(a, N);

    float *input, *output;
    hipMalloc(&input, sizeof(float) * N);
    hipMalloc(&output, sizeof(float) * block_num);
    hipMemcpy(input, a, sizeof(float) * N, hipMemcpyHostToDevice);

    dim3 block(thread_num_per_block);
    dim3 grid(block_num);

    reduce<<<grid, block>>>(input, output);

    hipMemcpy(res, output, sizeof(float) * block_num, hipMemcpyDeviceToHost);

    float sum = 0.0;
    for (int i = 0; i < block_num; i++)
        sum += res[i];
    cout << sum << endl;

    float hsum = 0.0;
    for (int i = 0; i < N; i++)
    {
        hsum += a[i];
    }
    cout << hsum << endl;

    hipFree(input);
    hipFree(output);
    delete[] a;
    delete[] res;
}