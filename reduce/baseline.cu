#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <iostream>
using namespace std;

const int N = 256 * 1024 * 128;
const int m = 1;
const int thread_num_per_block = 256;
const int block_num = N / (thread_num_per_block * m);

void initval(float s[], int n) {
    srand((unsigned)time(NULL));
    for (int i = 0; i < n; i++) {
        s[i] = float(rand() % 10) / 100000;
        // s[i] = 1.0;
    }
}

__global__ void reduce(float* input, float* output) {
    __shared__ float array[thread_num_per_block];
    int tid = threadIdx.x;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    array[tid] = input[id];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            array[tid] += array[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = array[0];
    }
}

int main(int argc, char** argv) {
    int dev = 3;
    hipSetDevice(dev);

    float* a = new float[N];
    float* res = new float[block_num];
    initval(a, N);

    float *input, *output;
    hipMalloc(&input, sizeof(float) * N);
    hipMalloc(&output, sizeof(float) * block_num);
    hipMemcpy(input, a, sizeof(float) * N, hipMemcpyHostToDevice);

    dim3 block(thread_num_per_block);
    dim3 grid(block_num);

    int iter = 1000;
    float msecTotal;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int i = 0; i < iter; i++) {
        reduce<<<grid, block>>>(input, output);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);

    hipMemcpy(res, output, sizeof(float) * block_num, hipMemcpyDeviceToHost);

    cout << "time:" << msecTotal / 1000 << "ms" << endl;

    // float sum = 0.0;
    // for (int i = 0; i < block_num; i++)
    //     sum += res[i];
    // cout << sum << endl;

    // float hsum = 0.0;
    // for (int i = 0; i < N; i++) {
    //     hsum += a[i];
    // }
    // cout << hsum << endl;

    hipFree(input);
    hipFree(output);
    delete[] a;
    delete[] res;
}